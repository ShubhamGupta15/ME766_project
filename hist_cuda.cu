#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <cmath>
using namespace std;

__global__ void create_histogram(int *hist, int *img, int *num_rows, int *num_cols){
	__shared__ int smallMatrix[3][3];
	__shared__ int decimal = 0;

	//each block handles one pixel in the image for histogram
	//hence each block has one small matrix

	int i = blockIdx.x;
	int j = blockIdx.y;
	int thx = threadIdx.x;

	if( img[i*200 j] < = img[(i - 1 + (thx / 3 ))*200 + j - 1 + (thx % 3)] ) {
		smallMatrix[(thx / 3 )][(thx % 3)] = 0;
	}
	else{
		smallMatrix[(thx / 3 )][(thx % 3)] = 1;
	}

	__syncthreads();

	if(threadIdx.x ==0){
		decimal = smallMatrix[0][0] * int(pow(2, 7)) + smallMatrix[0][1] * int(pow(2, 6)) + smallMatrix[0][2] * int(pow(2, 5)) +
		              smallMatrix[1][2] * int(pow(2, 4)) +
		              smallMatrix[2][2] * int(pow(2, 3)) + smallMatrix[2][1] * int(pow(2, 2)) + smallMatrix[2][0] * int(pow(2, 1)) +
		              smallMatrix[1][0] * 1;
	

		atomicAdd(*hist[decimal], 1);
	}

}


double distance(int * a, int *b, int size)
{
  double distance = 0;
  for (int i = 0; i < size; i ++) {
      if (a[i] + b[i] == 0) {
          distance += 0;
      }
      else {
          distance += 0.5 * pow ((a[i]- b[i]), 2) / (a[i] + b[i]);
      }
  }
   // printf("nbefore returning from distance function\n");
    return  distance;
}

int find_closest(int ***training_set, int num_persons, int num_training, int size, int * test_image)
{
  double ** dist = new double * [num_persons]; //make an array which will store the comparison values
    for (int i = 0; i < num_persons; i++) {
        dist[i] = new double [num_training];
    }
    for (int i = 0;  i < num_persons; i++) { //populate dhe distance array
        for (int j =0; j < num_training; j++) {
            dist[i][j] = distance(training_set[i][j], test_image, size);
        }
    }

    double closestValue = dist[0][0];
    int  closest = 1e9 ;

    for (int i = 0;  i < num_persons; i++) {
        for (int j =0; j < num_training; j++) {
            if (dist[i][j] < closestValue){
                closestValue = dist[i][j];
                closest = i;
        }
        }
    }
    for (int i = 0; i < num_persons; ++i) {
        delete dist[i];
    }
    delete []dist;
   // printf("before returning from find closest function\n");
    return closest + 1;
}

int **alloc_2d_matrix(int r, int c)
{
  int** a = new int*[r];
  for(int i = 0;i<r;i++)
  {
  	a[i] = new int[c];
  } 
  return a;
}

void dealloc_2d_matrix(int **a, int r, int c)
{
  for(int i = 0;i<r;i++)
  {
  	delete a[i];
  }
  delete [] a;
}

int ** read_image_data(string file_name, int h, int w)
{
  ifstream File;
    File.open(file_name);
    //cout << file_name << endl;
    
    int** data=alloc_2d_matrix(h,w);
    int tmp;
    for (int i = 0; i < h;i++) {
        for (int j = 0; j < w; j++) {
            File >> tmp;
            data[i][j] = tmp;
        //   cout <<data[i][j] ;

        }

    }

    File.close();
    return data;
}
int main()
{
    int nrOfIds = 9;
    int nrOfPhotosPerId = 5;
    int num_rows = 150;
    int num_cols = 200;
    int histogramSize = 256;
    int start_s=clock();
    int *hist, *d_img, *d_num_rows, *d_num_cols;


    int *** training_set = new int **[nrOfIds]; //nr of people, nr of images per person, histogram size
    for (int i = 0; i < nrOfIds; i++) {
        training_set[i] = alloc_2d_matrix(nrOfPhotosPerId,histogramSize);
    }

    for (int i = 0; i < nrOfIds; i++) { //initialize  training set to 0
        for (int j = 0; j < nrOfPhotosPerId; j++) {
            for (int e = 0; e < histogramSize; e++) {
                training_set[i][j][e] = 0;
            }
        }
    }

        //get file name

        string filename;
        for (int w = 1; w <= 9; w++) {
            for (int q = 1; q <= 5; q++) { //get all the file's names

                filename = "s0" + to_string(w) + "_0" + to_string(q) + "resized.txt";
                //cout<<filename<<endl;


                int **image = read_image_data(filename, num_rows, num_cols);
                int **img = alloc_2d_matrix((num_rows + 2), (num_cols + 2)); //enhanced image matrix with 0 in the corners

                for (int i = 0; i < (num_rows + 2); i++) { //initialize enhanced img matrix  0
                    for (int j = 0; j < (num_cols + 2); j++) {

                    	if(i==0 || j==0 || i == num_rows+1 || j == num_cols+1)  img[i][j] = 0;
                    	else img[i][j] = image[i - 1][j - 1];
                    }
                }

                hipMalloc(void** &hist, sizeof(int)*histogramSize);
                hipMalloc(void** &d_img, sizeof(int)*num_rows*num_cols);
                hipMalloc(void** &d_num_cols, sizeof(int));
                hipMalloc(void** &d_num_rows, sizeof(int));

                hipMemcpy(d_img, img[0], sizeof(int)*num_rows*num_cols, hipMemcpyHostToDevice);
                hipMemcpy(d_num_rows, num_rows, sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(d_num_rows, num_cols, sizeof(int), hipMemcpyHostToDevice);

				dim3 griddim(150,200);
				create_histogram<<<griddim,9>>>( hist, d_img, d_num_rows, d_num_cols);

                hipMemcpy(training_set[w - 1][q - 1], hist, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
                string err = hipGetErrorString(hipGetLastError ());
                cout<<err<<endl;
                hipFree(hist);
                hipFree(d_img);
                hipFree(d_num_rows);
                hipFree(d_num_cols);
                hipDeviceSynchronize();

                //deallocate images
                dealloc_2d_matrix(image, num_rows, num_cols);
                dealloc_2d_matrix(img, (num_rows + 2), (num_cols + 2));
            }
        }


        //TESTING PART
        string filenames;
        filenames = "s" + to_string(36) + "_0" + to_string(1) + "resized.txt";
                //cout<<filename<<endl;


        int **image11 = read_image_data(filenames, num_rows, num_cols);
        int **img11 = alloc_2d_matrix((num_rows + 2), (num_cols + 2)); //enhanced image matrix with 0 in the corners

        for (int i = 0; i < (num_rows + 2); i++) { //initialize enhanced img matrix  0
            for (int j = 0; j < (num_cols + 2); j++) {

            	if(i==0 || j==0 || i == num_rows+1 || j == num_cols+1)  img[i][j] = 0;
            	else img[i][j] = image[i - 1][j - 1];
            }
        }

        int *A = new int[256];
        for(int i =0; i< 256; i++){
            A[i] = 0;
        }

        hipMalloc(void** &hist, sizeof(int)*histogramSize);
                hipMalloc(void** &d_img, sizeof(int)*num_rows*num_cols);
                hipMalloc(void** &d_num_cols, sizeof(int));
                hipMalloc(void** &d_num_rows, sizeof(int));

                hipMemcpy(d_img, img11[0], sizeof(int)*num_rows*num_cols, hipMemcpyHostToDevice);
                hipMemcpy(d_num_rows, num_rows, sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(d_num_rows, num_cols, sizeof(int), hipMemcpyHostToDevice);

				dim3 griddim(150,200);
				create_histogram<<<griddim,9>>>( hist, d_img, d_num_rows, d_num_cols);

                hipMemcpy(A, hist, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
                string err = hipGetErrorString(hipGetLastError ());
                cout<<err<<endl;
                hipFree(hist);
                hipFree(d_img);
                hipFree(d_num_rows);
                hipFree(d_num_cols);
                hipDeviceSynchronize();

        for(int i =0; i<256; i++){
            //cout<<A[i]<<endl;
        }
        //deallocate images
        dealloc_2d_matrix(image11, num_rows, num_cols);
        dealloc_2d_matrix(img11, (num_rows + 2), (num_cols + 2));


        int testResultId;
        testResultId = find_closest(training_set,nrOfIds,nrOfPhotosPerId,histogramSize,A);
        cout<<testResultId<<endl;

        cout<<"Sucess!"<<endl;
        delete [] A;
}